#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <moderngpu/transform.hxx>
#include "commons.h"
#include "cudaCommons.h"

using namespace std;
using namespace mgpu;

__device__ int cuAbs( int i );

void CudaSimpleListRank(
    int *devRank, int N, int *devNext, int threadsPerBlockX, int blocksPerGridX, standard_context_t &context )
{
  int *notAllDone;
  hipHostMalloc( &notAllDone, sizeof( int ) );

  int *devTmp;
  int *devNotAllDone;

  CUCHECK( hipMalloc( (void **) &devTmp, sizeof( int ) * N ) );
  CUCHECK( hipMalloc( (void **) &devNotAllDone, sizeof( int ) ) );

  do
  {
    transform(
        [=] MGPU_DEVICE( int thid ) {
          if ( thid == 0 ) *devNotAllDone = 0;

          int nxt = devNext[thid];
          if ( nxt == -1 ) return;
          devTmp[thid] = devRank[nxt] + 1;
        },
        N,
        context );
    context.synchronize();

    transform(
        [=] MGPU_DEVICE( int thid ) {
          if ( devNext[thid] == -1 ) return;
          devRank[thid] += devTmp[thid];
        },
        N,
        context );
    context.synchronize();

    transform(
        [=] MGPU_DEVICE( int thid ) {
          if ( devNext[thid] == -1 ) return;
          devTmp[thid] = devNext[devNext[thid]];
        },
        N,
        context );
    context.synchronize();


    transform(
        [=] MGPU_DEVICE( int thid ) {
          if ( devNext[thid] == -1 ) return;

          devNext[thid] = devTmp[thid];

          *devNotAllDone = 1;
        },
        N,
        context );
    context.synchronize();

    CUCHECK( hipMemcpy( notAllDone, devNotAllDone, sizeof( int ), hipMemcpyDeviceToHost ) );
  } while ( *notAllDone );

  hipFree( notAllDone );
  CUCHECK( hipFree( devTmp ) );
  CUCHECK( hipFree( devNotAllDone ) );
}

void CudaFastListRank( int *devRank, int N, int head, int *devNext, standard_context_t &context )
{
  Timer listTimer( "List Rank" );
  int s;
  if ( N > 1000000 )
    s = 50000;
  else
    s = N / 100;
  if ( s == 0 ) s = 1;


  int *sum;
  int *last;
  int *next;
  sum = new int[s + 1];
  last = new int[s + 1];
  next = new int[N];
  // hipHostMalloc( &sum, sizeof( int ) * ( s + 1 ) );
  // hipHostMalloc( &last, sizeof( int ) * ( s + 1 ) );
  // hipHostMalloc( &next, sizeof( int ) * N );

  listTimer.measureTime( "Host Allocs" );

  int *devSum;
  CUCHECK( hipMalloc( (void **) &devSum, sizeof( int ) * ( s + 1 ) ) );
  int *devSublistHead;
  CUCHECK( hipMalloc( (void **) &devSublistHead, sizeof( int ) * ( s + 1 ) ) );
  int *devSublistId;
  CUCHECK( hipMalloc( (void **) &devSublistId, sizeof( int ) * N ) );
  int *devLast;
  CUCHECK( hipMalloc( (void **) &devLast, sizeof( int ) * ( s + 1 ) ) );

  listTimer.measureTime( "Device Allocs" );

  transform(
      [=] MGPU_DEVICE( int i ) {
        hiprandState state;
        hiprand_init( 123, i, 0, &state );

        int p = i * ( N / s );
        int q = min( p + N / s, N );

        int splitter;
        do
        {
          splitter = ( cuAbs( hiprand( &state ) ) % ( q - p ) ) + p;
        } while ( devNext[splitter] == -1 );

        devSublistHead[i + 1] = devNext[splitter];
        devNext[splitter] = -i - 2;  // To avoid confusion with -1

        if ( i == 0 )
        {
          devSublistHead[0] = head;
        }
      },
      s,
      context );

  listTimer.measureTime( "CPU generating splitters" );

  transform(
      [=] MGPU_DEVICE( int thid ) {
        int current;
        current = devSublistHead[thid];

        int counter = 0;
        while ( current >= 0 )
        {
          devRank[current] = counter;
          counter++;

          int n = devNext[current];

          if ( n < 0 )
          {
            devSum[thid] = counter;
            devLast[thid] = current;
          }

          devSublistId[current] = thid;
          current = n;
        }
      },
      s + 1,
      context );
  context.synchronize();

  listTimer.measureTime( "GPU sublist rank calculation" );

  CUCHECK( hipMemcpy( sum, devSum, sizeof( int ) * ( s + 1 ), hipMemcpyDeviceToHost ) );
  CUCHECK( hipMemcpy( next, devNext, sizeof( int ) * N, hipMemcpyDeviceToHost ) );
  CUCHECK( hipMemcpy( last, devLast, sizeof( int ) * ( s + 1 ), hipMemcpyDeviceToHost ) );

  listTimer.measureTime( "Copy sublists to Host" );


  int tmpSum = 0;
  int current = head;
  int currentSublist = 0;
  for ( int i = 0; i <= s; i++ )
  {
    tmpSum += sum[currentSublist];
    sum[currentSublist] = tmpSum - sum[currentSublist];

    current = last[currentSublist];
    currentSublist = -next[current] - 1;
  }


  CUCHECK( hipMemcpy( devSum, sum, sizeof( int ) * ( s + 1 ), hipMemcpyHostToDevice ) );

  listTimer.measureTime( "CPU adding sums" );

  transform(
      [=] MGPU_DEVICE( int thid ) {
        int sublistId = devSublistId[thid];
        devRank[thid] += devSum[sublistId];
      },
      N,
      context );
  context.synchronize();

  listTimer.measureTime( "GPU final rank" );

  delete[] next;
  delete[] sum;
  delete[] last;
  // hipHostFree( next );
  // hipHostFree( sum );
  // hipHostFree( last );

  CUCHECK( hipFree( devSum ) );
  CUCHECK( hipFree( devSublistHead ) );
  CUCHECK( hipFree( devSublistId ) );
  CUCHECK( hipFree( devLast ) );

  listTimer.measureTime( "Free moemory" );
  listTimer.setPrefix( "" );
}

void CudaAssert( hipError_t error, const char *code, const char *file, int line )
{
  if ( error != hipSuccess )
  {
    cerr << "Cuda error :" << code << ", " << file << ":" << error << endl;
    exit( 1 );
  }
}

void CudaPrintTab( int *tab, int size )
{
  int *tmp = (int *) malloc( sizeof( int ) * size );
  CUCHECK( hipMemcpy( tmp, tab, sizeof( int ) * size, hipMemcpyDeviceToHost ) );

  for ( int i = 0; i < size; i++ )
  {
    cerr << tmp[i] << " ";
  }
  cerr << endl;

  free( tmp );
}
__device__ int cuAbs( int i )
{
  return i < 0 ? -i : i;
}