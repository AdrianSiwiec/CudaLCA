#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <moderngpu/transform.hxx>
#include "commons.h"
#include "cudaCommons.h"

#define ull unsigned long long

using namespace std;
using namespace mgpu;

__device__ int cuAbs( int i );

void CudaSimpleListRank(
    int *devRank, int N, int *devNext, int threadsPerBlockX, int blocksPerGridX, standard_context_t &context )
{
  int *notAllDone;
  hipHostMalloc( &notAllDone, sizeof( int ) );

  ull *devRankNext;
  int *devTmpRank;
  int *devTmpNext;
  int *devNotAllDone;

  CUCHECK( hipMalloc( (void **) &devRankNext, sizeof( ull ) * N ) );
  CUCHECK( hipMalloc( (void **) &devTmpRank, sizeof( int ) * N ) );
  CUCHECK( hipMalloc( (void **) &devTmpNext, sizeof( int ) * N ) );
  CUCHECK( hipMalloc( (void **) &devNotAllDone, sizeof( int ) ) );

  transform(
      [=] MGPU_DEVICE( int thid ) { devRankNext[thid] = ( ( (ull) 0 ) << 32 ) + devNext[thid] + 1; }, N, context );

  const int loopsWithoutSync = 10;

  do
  {
    transform(
        [=] MGPU_DEVICE( int thid ) {
          ull rankNext = devRankNext[thid];
          for ( int i = 0; i < loopsWithoutSync; i++ )
          {
            if ( thid == 0 ) *devNotAllDone = 0;

            int rank = rankNext >> 32;
            int nxt = rankNext - 1;

            if ( nxt != -1 )
            {
              ull grandNext = devRankNext[nxt];

              rank += ( grandNext >> 32 ) + 1;
              nxt = grandNext - 1;

              rankNext = ( ( (ull) rank ) << 32 ) + nxt + 1;
              atomicExch( devRankNext + thid, rankNext );

              if ( i == loopsWithoutSync - 1 ) *devNotAllDone = 1;
            }
          }
        },
        N,
        context );

    context.synchronize();

    CUCHECK( hipMemcpy( notAllDone, devNotAllDone, sizeof( int ), hipMemcpyDeviceToHost ) );
  } while ( *notAllDone );

  transform( [=] MGPU_DEVICE( int thid ) { devRank[thid] = devRankNext[thid] >> 32; }, N, context );
  context.synchronize();

  hipFree( notAllDone );
  CUCHECK( hipFree( devTmpRank ) );
  CUCHECK( hipFree( devTmpNext ) );
  CUCHECK( hipFree( devNotAllDone ) );
}

void CudaFastListRank( int *devRank, int N, int head, int *devNext, standard_context_t &context )
{
  Timer listTimer( "List Rank" );
  int s;
  if ( N > 1000000 )
    s = 50000;
  else
    s = N / 100;
  if ( s == 0 ) s = 1;


  int *sum;
  int *last;
  int *next;
  sum = new int[s + 1];
  last = new int[s + 1];
  next = new int[N];
  // hipHostMalloc( &sum, sizeof( int ) * ( s + 1 ) );
  // hipHostMalloc( &last, sizeof( int ) * ( s + 1 ) );
  // hipHostMalloc( &next, sizeof( int ) * N );

  listTimer.measureTime( "Host Allocs" );

  int *devSum;
  CUCHECK( hipMalloc( (void **) &devSum, sizeof( int ) * ( s + 1 ) ) );
  int *devSublistHead;
  CUCHECK( hipMalloc( (void **) &devSublistHead, sizeof( int ) * ( s + 1 ) ) );
  int *devSublistId;
  CUCHECK( hipMalloc( (void **) &devSublistId, sizeof( int ) * N ) );
  int *devLast;
  CUCHECK( hipMalloc( (void **) &devLast, sizeof( int ) * ( s + 1 ) ) );

  listTimer.measureTime( "Device Allocs" );

  transform(
      [=] MGPU_DEVICE( int i ) {
        hiprandState state;
        hiprand_init( 123, i, 0, &state );

        int p = i * ( N / s );
        int q = min( p + N / s, N );

        int splitter;
        do
        {
          splitter = ( cuAbs( hiprand( &state ) ) % ( q - p ) ) + p;
        } while ( devNext[splitter] == -1 );

        devSublistHead[i + 1] = devNext[splitter];
        devNext[splitter] = -i - 2;  // To avoid confusion with -1

        if ( i == 0 )
        {
          devSublistHead[0] = head;
        }
      },
      s,
      context );

  listTimer.measureTime( "CPU generating splitters" );

  transform(
      [=] MGPU_DEVICE( int thid ) {
        int current;
        current = devSublistHead[thid];

        int counter = 0;
        while ( current >= 0 )
        {
          devRank[current] = counter;
          counter++;

          int n = devNext[current];

          if ( n < 0 )
          {
            devSum[thid] = counter;
            devLast[thid] = current;
          }

          devSublistId[current] = thid;
          current = n;
        }
      },
      s + 1,
      context );
  context.synchronize();

  listTimer.measureTime( "GPU sublist rank calculation" );

  CUCHECK( hipMemcpy( sum, devSum, sizeof( int ) * ( s + 1 ), hipMemcpyDeviceToHost ) );
  CUCHECK( hipMemcpy( next, devNext, sizeof( int ) * N, hipMemcpyDeviceToHost ) );
  CUCHECK( hipMemcpy( last, devLast, sizeof( int ) * ( s + 1 ), hipMemcpyDeviceToHost ) );

  listTimer.measureTime( "Copy sublists to Host" );


  int tmpSum = 0;
  int current = head;
  int currentSublist = 0;
  for ( int i = 0; i <= s; i++ )
  {
    tmpSum += sum[currentSublist];
    sum[currentSublist] = tmpSum - sum[currentSublist];

    current = last[currentSublist];
    currentSublist = -next[current] - 1;
  }


  CUCHECK( hipMemcpy( devSum, sum, sizeof( int ) * ( s + 1 ), hipMemcpyHostToDevice ) );

  listTimer.measureTime( "CPU adding sums" );

  transform(
      [=] MGPU_DEVICE( int thid ) {
        int sublistId = devSublistId[thid];
        devRank[thid] += devSum[sublistId];
      },
      N,
      context );
  context.synchronize();

  listTimer.measureTime( "GPU final rank" );

  delete[] next;
  delete[] sum;
  delete[] last;
  // hipHostFree( next );
  // hipHostFree( sum );
  // hipHostFree( last );

  CUCHECK( hipFree( devSum ) );
  CUCHECK( hipFree( devSublistHead ) );
  CUCHECK( hipFree( devSublistId ) );
  CUCHECK( hipFree( devLast ) );

  listTimer.measureTime( "Free moemory" );
  listTimer.setPrefix( "" );
}

void CudaAssert( hipError_t error, const char *code, const char *file, int line )
{
  if ( error != hipSuccess )
  {
    cerr << "Cuda error :" << code << ", " << file << ":" << error << endl;
    exit( 1 );
  }
}

void CudaPrintTab( int *tab, int size )
{
  int *tmp = (int *) malloc( sizeof( int ) * size );
  CUCHECK( hipMemcpy( tmp, tab, sizeof( int ) * size, hipMemcpyDeviceToHost ) );

  for ( int i = 0; i < size; i++ )
  {
    cerr << tmp[i] << " ";
  }
  cerr << endl;

  free( tmp );
}
__device__ int cuAbs( int i )
{
  return i < 0 ? -i : i;
}