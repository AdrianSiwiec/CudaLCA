#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "cudaCommons.h"

using namespace std;

__global__ void cuCalcRankRead( int V, int *next, int *depth, int *tmp, int *notAllDone );
__global__ void cuCalcRankWrite( int V, int *next, int *depth, int *tmp );
__global__ void cuMoveNextRead( int V, int *next, int *tmp );
__global__ void cuMoveNextWrite( int V, int *next, int *tmp, int *notAllDone );

void CudaListRank( int *devRank, int N, int *devNext, int threadsPerBlockX, int blocksPerGridX )
{
  int *notAllDone = (int *) malloc( sizeof( int ) );

  int *devTmp;
  int *devNotAllDone;

  CUCHECK( hipMalloc( (void **) &devTmp, sizeof( int ) * N ) );
  CUCHECK( hipMalloc( (void **) &devNotAllDone, sizeof( int ) ) );

  do
  {
    cuCalcRankRead<<<blocksPerGridX, threadsPerBlockX>>>( N, devNext, devRank, devTmp, devNotAllDone );
    CUCHECK( hipDeviceSynchronize() );

    cuCalcRankWrite<<<blocksPerGridX, threadsPerBlockX>>>( N, devNext, devRank, devTmp );
    CUCHECK( hipDeviceSynchronize() );

    cuMoveNextRead<<<blocksPerGridX, threadsPerBlockX>>>( N, devNext, devTmp );
    CUCHECK( hipDeviceSynchronize() );

    cuMoveNextWrite<<<blocksPerGridX, threadsPerBlockX>>>( N, devNext, devTmp, devNotAllDone );
    CUCHECK( hipDeviceSynchronize() );

    CUCHECK( hipMemcpy( notAllDone, devNotAllDone, sizeof( int ), hipMemcpyDeviceToHost ) );
  } while ( *notAllDone );

  free( notAllDone );
  CUCHECK( hipFree( devTmp ) );
  CUCHECK( hipFree( devNotAllDone ) );
}

__global__ void cuCalcRankRead( int V, int *next, int *rank, int *tmp, int *notAllDone )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
  if ( thid == 0 ) *notAllDone = 0;

  if ( thid >= V || next[thid] == -1 ) return;

  tmp[thid] = rank[next[thid]] + 1;
}

__global__ void cuCalcRankWrite( int V, int *next, int *rank, int *tmp )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( thid >= V || next[thid] == -1 ) return;

  rank[thid] += tmp[thid];
}

__global__ void cuMoveNextRead( int V, int *next, int *tmp )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( thid >= V || next[thid] == -1 ) return;

  tmp[thid] = next[next[thid]];
}

__global__ void cuMoveNextWrite( int V, int *next, int *tmp, int *notAllDone )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( thid >= V || next[thid] == -1 ) return;

  next[thid] = tmp[thid];

  *notAllDone = 1;
}

void CudaAssert( hipError_t error, const char *code, const char *file, int line )
{
  if ( error != hipSuccess )
  {
        cerr << "Cuda error :" << code << ", " << file << ":" << error << endl;
    exit( 1 );
  }
}

void CudaPrintTab( int *tab, int size )
{
  int *tmp = (int *) malloc( sizeof( int ) * size );
  CUCHECK( hipMemcpy( tmp, tab, sizeof( int ) * size, hipMemcpyDeviceToHost ) );

  for ( int i = 0; i < size; i++ )
  {
    cerr << tmp[i] << " ";
  }
  cerr << endl;

  free( tmp );
}