#include "hip/hip_runtime.h"
#include <iostream>
#include "commons.h"
#include "cudaCommons.h"

using namespace std;

__global__ void hipInit( int V, int *father, int *next, int *depth );
__global__ void cuCalcDepthRead( int V, int *next, int *depth, int *tmp, int *notAllDone );
__global__ void cuCalcDepthWrite( int V, int *next, int *depth, int *tmp );
__global__ void cuMoveNextRead( int V, int *next, int *tmp );
__global__ void cuMoveNextWrite( int V, int *next, int *tmp, int *notAllDone );
__global__ void cuCalcQueries( int Q, int *father, int *depth, int *queries, int *answers );

int main( int argc, char *argv[] )
{
  Timer timer = Timer();

  TestCase tc;
  if ( argc == 1 )
  {
    tc = readFromStdIn();
  }
  else
  {
    tc = readFromFile( argv[1] );
  }

  timer.measureTime( "Read Input" );

  int *devFather;
  int *devDepth;
  int *devNext;
  int *devNotAllDone;
  int *devQueries;
  int *devAnswers;
  int *devTmp;

  const int V = tc.tree.V;

  CUCHECK( hipMalloc( (void **) &devFather, sizeof( int ) * V ) );
  CUCHECK( hipMalloc( (void **) &devDepth, sizeof( int ) * V ) );
  CUCHECK( hipMalloc( (void **) &devNext, sizeof( int ) * V ) );
  CUCHECK( hipMalloc( (void **) &devTmp, sizeof( int ) * V ) );

  CUCHECK( hipMalloc( (void **) &devNotAllDone, sizeof( int ) ) );

  timer.measureTime( "Cuda Allocs" );

  CUCHECK( hipMemcpy( devFather, tc.tree.father.data(), sizeof( int ) * V, hipMemcpyHostToDevice ) );

  int threadsPerBlockX = 1024;
  int blockPerGridX = ( V + threadsPerBlockX - 1 ) / threadsPerBlockX;

  hipInit<<<blockPerGridX, threadsPerBlockX>>>( V, devFather, devNext, devDepth );
  CUCHECK( hipDeviceSynchronize() );

  timer.measureTime( "Copy Input and Init data" );

  int *notAllDone = (int *) malloc( sizeof( int ) );
  do
  {
    cuCalcDepthRead<<<blockPerGridX, threadsPerBlockX>>>( V, devNext, devDepth, devTmp, devNotAllDone );
    CUCHECK( hipDeviceSynchronize() );

    cuCalcDepthWrite<<<blockPerGridX, threadsPerBlockX>>>( V, devNext, devDepth, devTmp );
    CUCHECK( hipDeviceSynchronize() );

    cuMoveNextRead<<<blockPerGridX, threadsPerBlockX>>>( V, devNext, devTmp );
    CUCHECK( hipDeviceSynchronize() );

    cuMoveNextWrite<<<blockPerGridX, threadsPerBlockX>>>( V, devNext, devTmp, devNotAllDone );
    CUCHECK( hipDeviceSynchronize() );

    CUCHECK( hipMemcpy( notAllDone, devNotAllDone, sizeof( int ), hipMemcpyDeviceToHost ) );

  } while ( *notAllDone );

  timer.measureTime( "Cuda Preprocessing" );

  // int *depth = (int *) malloc( sizeof( int ) * V );

  // res = hipMemcpyDtoH( depth, devDepth, sizeof( int ) * V );
  // testRes( res, "Copy devDepth to host" );

  //   for ( int i = 0; i < V; i++ )
  //   {
  //     cout << i << ": " << depth[i] << endl;
  //   }

  int Q = tc.q.N;

  CUCHECK( hipMalloc( (void **) &devQueries, sizeof( int ) * Q * 2 ) );
  CUCHECK( hipMalloc( (void **) &devAnswers, sizeof( int ) * Q ) );

  CUCHECK( hipMemcpy( devQueries, tc.q.tab.data(), sizeof( int ) * Q * 2, hipMemcpyHostToDevice ) );

  timer.measureTime( "Copy Queries to Dev" );

  blockPerGridX = ( Q + threadsPerBlockX - 1 ) / threadsPerBlockX;

  cuCalcQueries<<<blockPerGridX, threadsPerBlockX>>>( Q, devFather, devDepth, devQueries, devAnswers );
  CUCHECK( hipDeviceSynchronize() );

  timer.measureTime( "Cuda calc queries" );

  int *answers = (int *) malloc( sizeof( int ) * Q );

  CUCHECK( hipMemcpy( answers, devAnswers, sizeof( int ) * Q, hipMemcpyDeviceToHost ) );

  timer.measureTime( "Copy answers to Host" );

  if ( argc < 3 )
  {
    writeAnswersToStdOut( Q, answers );
  }
  else
  {
    writeAnswersToFile( Q, answers, argv[2] );
  }

  timer.measureTime( "Write Output" );
}

__global__ void hipInit( int V, int *father, int *next, int *depth )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( thid >= V ) return;

  next[thid] = father[thid];
  depth[thid] = 0;
}

__global__ void cuCalcDepthRead( int V, int *next, int *depth, int *tmp, int *notAllDone )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
  if ( thid == 0 ) *notAllDone = 0;

  if ( thid >= V || next[thid] == -1 ) return;

  tmp[thid] = depth[next[thid]] + 1;
}

__global__ void cuCalcDepthWrite( int V, int *next, int *depth, int *tmp )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( thid >= V || next[thid] == -1 ) return;

  depth[thid] += tmp[thid];
}

__global__ void cuMoveNextRead( int V, int *next, int *tmp )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( thid >= V || next[thid] == -1 ) return;

  tmp[thid] = next[next[thid]];
}

__global__ void cuMoveNextWrite( int V, int *next, int *tmp, int *notAllDone )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( thid >= V || next[thid] == -1 ) return;

  next[thid] = tmp[thid];

  *notAllDone = 1;
}

__global__ void cuCalcQueries( int Q, int *father, int *depth, int *queries, int *answers )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( thid >= Q ) return;

  int p = queries[thid * 2];
  int q = queries[thid * 2 + 1];

  if ( p == q ) answers[thid] = p;

  while ( depth[p] != depth[q] )
  {
    if ( depth[p] > depth[q] )
      p = father[p];
    else
      q = father[q];
  }

  while ( p != q )
  {
    p = father[p];
    q = father[q];
  }

  answers[thid] = p;
}