#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

void CudaAssert( hipError_t error, const char* code, const char* file, int line )
{
  if ( error != hipSuccess )
  {
    cerr << "Cuda error :" << code << ", " << file << ":" << endl;
    exit( 1 );
  }
}