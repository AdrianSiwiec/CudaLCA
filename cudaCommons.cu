#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <iostream>
#include <moderngpu/transform.hxx>
#include "commons.h"
#include "cudaCommons.h"

#define ull unsigned long long

using namespace std;
using namespace mgpu;

__device__ int cuAbs( int i );

void CudaSimpleListRank( int *devRank, int N, int *devNext, standard_context_t &context )
{
  int *notAllDone;
  hipHostMalloc( &notAllDone, sizeof( int ) );

  ull *devRankNext;
  int *devNotAllDone;

  CUCHECK( hipMalloc( (void **) &devRankNext, sizeof( ull ) * N ) );
  CUCHECK( hipMalloc( (void **) &devNotAllDone, sizeof( int ) ) );

  transform(
      [=] MGPU_DEVICE( int thid ) { devRankNext[thid] = ( ( (ull) 0 ) << 32 ) + devNext[thid] + 1; }, N, context );

  const int loopsWithoutSync = 10;

  do
  {
    transform(
        [=] MGPU_DEVICE( int thid ) {
          ull rankNext = devRankNext[thid];
          for ( int i = 0; i < loopsWithoutSync; i++ )
          {
            if ( thid == 0 ) *devNotAllDone = 0;

            int rank = rankNext >> 32;
            int nxt = rankNext - 1;

            if ( nxt != -1 )
            {
              ull grandNext = devRankNext[nxt];

              rank += ( grandNext >> 32 ) + 1;
              nxt = grandNext - 1;

              rankNext = ( ( (ull) rank ) << 32 ) + nxt + 1;
              atomicExch( devRankNext + thid, rankNext );

              if ( i == loopsWithoutSync - 1 ) *devNotAllDone = 1;
            }
          }
        },
        N,
        context );

    context.synchronize();

    CUCHECK( hipMemcpy( notAllDone, devNotAllDone, sizeof( int ), hipMemcpyDeviceToHost ) );
  } while ( *notAllDone );

  transform( [=] MGPU_DEVICE( int thid ) { devRank[thid] = devRankNext[thid] >> 32; }, N, context );
  context.synchronize();

  hipFree( notAllDone );
  hipFree( devRankNext );
  CUCHECK( hipFree( devNotAllDone ) );
}

void CudaFastListRank( int *devRank, int N, int head, int *devNext, standard_context_t &context )
{
  Timer listTimer( "List Rank" );
  int s;
  if ( N > 1000000 )
  {
    double tmp = N;
    tmp /= log2( N );
    tmp /= log2( N );
    s = tmp;
  }
  else
    s = N / 100;
  if ( s == 0 ) s = 1;

  cerr << s << endl;


  int *devSum;
  CUCHECK( hipMalloc( (void **) &devSum, sizeof( int ) * ( s + 1 ) ) );
  int *devSublistHead;
  CUCHECK( hipMalloc( (void **) &devSublistHead, sizeof( int ) * ( s + 1 ) ) );
  int *devSublistId;
  CUCHECK( hipMalloc( (void **) &devSublistId, sizeof( int ) * N ) );
  int *devLast;
  CUCHECK( hipMalloc( (void **) &devLast, sizeof( int ) * ( s + 1 ) ) );

  listTimer.measureTime( "Device Allocs" );

  transform(
      [=] MGPU_DEVICE( int i ) {
        hiprandState state;
        hiprand_init( 123, i, 0, &state );

        int p = i * ( N / s );
        int q = min( p + N / s, N );

        int splitter;
        do
        {
          splitter = ( cuAbs( hiprand( &state ) ) % ( q - p ) ) + p;
        } while ( devNext[splitter] == -1 );

        devSublistHead[i + 1] = devNext[splitter];
        devNext[splitter] = -i - 2;  // To avoid confusion with -1

        if ( i == 0 )
        {
          devSublistHead[0] = head;
        }
      },
      s,
      context );

  context.synchronize();
  listTimer.measureTime( "GPU generating splitters" );

  transform(
      [=] MGPU_DEVICE( int thid ) {
        int current = devSublistHead[thid];
        int counter = 0;

        while ( current >= 0 )
        {
          devRank[current] = counter++;

          int n = devNext[current];

          if ( n < 0 )
          {
            devSum[thid] = counter;
            devLast[thid] = current;
          }

          devSublistId[current] = thid;
          current = n;
        }
      },
      s + 1,
      context );
  context.synchronize();

  listTimer.measureTime( "GPU sublist rank calculation" );

  transform(
      [=] MGPU_DEVICE( int thid ) {
        int tmpSum = 0;
        int current = head;
        int currentSublist = 0;
        for ( int i = 0; i <= s; i++ )
        {
          tmpSum += devSum[currentSublist];
          devSum[currentSublist] = tmpSum - devSum[currentSublist];

          current = devLast[currentSublist];
          currentSublist = -devNext[current] - 1;
        }
      },
      1,
      context );

  context.synchronize();
  listTimer.measureTime( "GPU Adding Times" );

  transform(
      [=] MGPU_DEVICE( int thid ) {
        int sublistId = devSublistId[thid];
        devRank[thid] += devSum[sublistId];
      },
      N,
      context );
  context.synchronize();

  listTimer.measureTime( "GPU final rank" );

  CUCHECK( hipFree( devSum ) );
  CUCHECK( hipFree( devSublistHead ) );
  CUCHECK( hipFree( devSublistId ) );
  CUCHECK( hipFree( devLast ) );

  listTimer.measureTime( "Free moemory" );
  listTimer.setPrefix( "" );
}

void CudaAssert( hipError_t error, const char *code, const char *file, int line )
{
  if ( error != hipSuccess )
  {
    cerr << "Cuda error :" << code << ", " << file << ":" << error << endl;
    exit( 1 );
  }
}

void CudaPrintTab( int *tab, int size )
{
  int *tmp = (int *) malloc( sizeof( int ) * size );
  CUCHECK( hipMemcpy( tmp, tab, sizeof( int ) * size, hipMemcpyDeviceToHost ) );

  for ( int i = 0; i < size; i++ )
  {
    cerr << tmp[i] << " ";
  }
  cerr << endl;

  free( tmp );
}
__device__ int cuAbs( int i )
{
  return i < 0 ? -i : i;
}