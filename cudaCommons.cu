#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <moderngpu/transform.hxx>
#include "commons.h"
#include "cudaCommons.h"

using namespace std;
using namespace mgpu;

__global__ void cuCalcRankRead( int V, int *next, int *depth, int *tmp, int *notAllDone );
__global__ void cuCalcRankWrite( int V, int *next, int *depth, int *tmp );
__global__ void cuMoveNextRead( int V, int *next, int *tmp );
__global__ void cuMoveNextWrite( int V, int *next, int *tmp, int *notAllDone );

void CudaSimpleListRank( int *devRank, int N, int *devNext, int threadsPerBlockX, int blocksPerGridX )
{
  int *notAllDone = (int *) malloc( sizeof( int ) );

  int *devTmp;
  int *devNotAllDone;

  CUCHECK( hipMalloc( (void **) &devTmp, sizeof( int ) * N ) );
  CUCHECK( hipMalloc( (void **) &devNotAllDone, sizeof( int ) ) );

  do
  {
    cuCalcRankRead<<<blocksPerGridX, threadsPerBlockX>>>( N, devNext, devRank, devTmp, devNotAllDone );
    CUCHECK( hipDeviceSynchronize() );

    cuCalcRankWrite<<<blocksPerGridX, threadsPerBlockX>>>( N, devNext, devRank, devTmp );
    CUCHECK( hipDeviceSynchronize() );

    cuMoveNextRead<<<blocksPerGridX, threadsPerBlockX>>>( N, devNext, devTmp );
    CUCHECK( hipDeviceSynchronize() );

    cuMoveNextWrite<<<blocksPerGridX, threadsPerBlockX>>>( N, devNext, devTmp, devNotAllDone );
    CUCHECK( hipDeviceSynchronize() );

    CUCHECK( hipMemcpy( notAllDone, devNotAllDone, sizeof( int ), hipMemcpyDeviceToHost ) );
  } while ( *notAllDone );

  free( notAllDone );
  CUCHECK( hipFree( devTmp ) );
  CUCHECK( hipFree( devNotAllDone ) );
}

void CudaFastListRank( int *devRank, int N, int head, int *devNext, standard_context_t &context )
{
  Timer listTimer( "List Rank" );
  int s;
  if ( N > 1000000 )
    s = 1000;
  else
    s = N / 100;
  if ( s == 0 ) s = 1;


  int *next = new int[N];
  int *sum = new int[s + 1];
  int *last = new int[s + 1];
  int *sublistHead = new int[s + 1];

  int *devSum;
  CUCHECK( hipMalloc( (void **) &devSum, sizeof( int ) * ( s + 1 ) ) );
  int *devSublistHead;
  CUCHECK( hipMalloc( (void **) &devSublistHead, sizeof( int ) * ( s + 1 ) ) );
  int *devSublistId;
  CUCHECK( hipMalloc( (void **) &devSublistId, sizeof( int ) * N ) );
  int *devLast;
  CUCHECK( hipMalloc( (void **) &devLast, sizeof( int ) * ( s + 1 ) ) );


  CUCHECK( hipMemcpy( next, devNext, sizeof( int ) * N, hipMemcpyDeviceToHost ) );

  for ( int i = 0; i < s; i++ )
  {
    int p = i * ( N / s );
    int q = min( p + N / s, N );

    int splitter;
    do
    {
      splitter = ( rand() % ( q - p ) ) + p;
    } while ( next[splitter] == -1 );

    sublistHead[i + 1] = next[splitter];
    next[splitter] = -i - 2;  // To avoid confusion with -1
  }
  sublistHead[0] = head;

  CUCHECK( hipMemcpy( devNext, next, sizeof( int ) * N, hipMemcpyHostToDevice ) );

  CUCHECK( hipMemcpy( devSublistHead, sublistHead, sizeof( int ) * ( s + 1 ), hipMemcpyHostToDevice ) );

  listTimer.measureTime( "Init and CPU generating splitters" );

  transform(
      [=] MGPU_DEVICE( int thid ) {
        int current;
        current = devSublistHead[thid];

        int counter = 0;
        while ( current >= 0 )
        {
          devRank[current] = counter;
          counter++;

          int n = devNext[current];

          if ( n < 0 )
          {
            devSum[thid] = counter;
            devLast[thid] = current;
          }

          devSublistId[current] = thid;
          current = n;
        }
      },
      s + 1,
      context );

  listTimer.measureTime( "GPU sublist rank calculation" );


  CUCHECK( hipMemcpy( sum, devSum, sizeof( int ) * ( s + 1 ), hipMemcpyDeviceToHost ) );
  CUCHECK( hipMemcpy( next, devNext, sizeof( int ) * N, hipMemcpyDeviceToHost ) );

  CUCHECK( hipMemcpy( last, devLast, sizeof( int ) * ( s + 1 ), hipMemcpyDeviceToHost ) );


  int tmpSum = 0;
  int current = head;
  int currentSublist = 0;
  for ( int i = 0; i <= s; i++ )
  {
    tmpSum += sum[currentSublist];
    sum[currentSublist] = tmpSum - sum[currentSublist];

    current = last[currentSublist];
    currentSublist = -next[current] - 1;
  }


  CUCHECK( hipMemcpy( devSum, sum, sizeof( int ) * ( s + 1 ), hipMemcpyHostToDevice ) );

  listTimer.measureTime( "CPU adding sums and copying memory" );

  transform(
      [=] MGPU_DEVICE( int thid ) {
        int sublistId = devSublistId[thid];
        devRank[thid] += devSum[sublistId];
      },
      N,
      context );
  context.synchronize();

  listTimer.measureTime( "GPU final rank" );

  delete[] next;
  delete[] sum;
  delete[] last;
  delete[] sublistHead;

  CUCHECK( hipFree( devSum ) );
  CUCHECK( hipFree( devSublistHead ) );
  CUCHECK( hipFree( devSublistId ) );
  CUCHECK( hipFree( devLast ) );

  listTimer.measureTime( "Free moemory" );
  listTimer.setPrefix("");
}

__global__ void cuCalcRankRead( int V, int *next, int *rank, int *tmp, int *notAllDone )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;
  if ( thid == 0 ) *notAllDone = 0;

  if ( thid >= V || next[thid] == -1 ) return;

  tmp[thid] = rank[next[thid]] + 1;
}

__global__ void cuCalcRankWrite( int V, int *next, int *rank, int *tmp )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( thid >= V || next[thid] == -1 ) return;

  rank[thid] += tmp[thid];
}

__global__ void cuMoveNextRead( int V, int *next, int *tmp )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( thid >= V || next[thid] == -1 ) return;

  tmp[thid] = next[next[thid]];
}

__global__ void cuMoveNextWrite( int V, int *next, int *tmp, int *notAllDone )
{
  int thid = ( blockIdx.x * blockDim.x ) + threadIdx.x;

  if ( thid >= V || next[thid] == -1 ) return;

  next[thid] = tmp[thid];

  *notAllDone = 1;
}

void CudaAssert( hipError_t error, const char *code, const char *file, int line )
{
  if ( error != hipSuccess )
  {
    cerr << "Cuda error :" << code << ", " << file << ":" << error << endl;
    exit( 1 );
  }
}

void CudaPrintTab( int *tab, int size )
{
  int *tmp = (int *) malloc( sizeof( int ) * size );
  CUCHECK( hipMemcpy( tmp, tab, sizeof( int ) * size, hipMemcpyDeviceToHost ) );

  for ( int i = 0; i < size; i++ )
  {
    cerr << tmp[i] << " ";
  }
  cerr << endl;

  free( tmp );
}